#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

//-----------------------------------------------------------------------------
// GpuConstantsPackage: a struct to hold many constants (including pointers
//                      to allocated memory on the device) that can be
//                      uploaded all at once.  Placing this in the "constants
//                      cache" is a convenient and performant way of handling
//                      constant information on the GPU.
//-----------------------------------------------------------------------------
struct GpuConstantsPackage {
  int     nparticle;
  int*    partType;
  float*  partX;
  float*  partY;
  float*  partZ;
  float*  partQ;
  float*  Etot;
};
typedef struct GpuConstantsPackage cribSheet;

// This device constant is available to all functions in this CUDA unit
__device__ __constant__ cribSheet cSh;

//-----------------------------------------------------------------------------
// GpuMirroredInt: a struct holding mirrored int data on both the CPU and the
//                 GPU.  Functions below will operate on this struct
//                 (because this isn't a workshop on C++)
//-----------------------------------------------------------------------------
struct GpuMirroredInt {
  int len;          // Length of the array (again, this is not a C++ course)
  int IsPinned;     // "Pinned" memory is best for Host <= => GPU transfers.
                    //   In fact, if non-pinned memory is transferred to the
                    //   GPU from the host, a temporary allocation of pinned
                    //   memory will be created and then destroyed.  Pinned
                    //   memory is not host-pageable, but the only performance
                    //   implication is that creating lots of pinned memory
                    //   may make it harder for the host OS to manage large
                    //   memory jobs.
  int* HostData;    // Pointer to allocated memory on the host
  int* DevcData;    // Pointer to allocated memory on the GPU.  Note that the
                    //   host can know what the address of memory on the GPU
                    //   is, but it cannot simply de-reference that pointer
                    //   in host code.
};
typedef struct GpuMirroredInt gpuInt;

//-----------------------------------------------------------------------------
// GpuMirroredInt: a struct holding mirrored fp32 data on both the CPU and the
//                 GPU.  Functions below will operate on this struct
//                 (because this isn't a workshop on C++)
//-----------------------------------------------------------------------------
struct GpuMirroredFloat {
  int len;          // Length of the array (again, this is not a C++ course)
  int IsPinned;     // "Pinned" memory is best for Host <= => GPU transfers.
                    //   In fact, if non-pinned memory is transferred to the
                    //   GPU from the host, a temporary allocation of pinned
                    //   memory will be created and then destroyed.  Pinned
                    //   memory is not host-pageable, but the only performance
                    //   implication is that creating lots of pinned memory
                    //   may make it harder for the host OS to manage large
                    //   memory jobs.
  float* HostData;  // Pointer to allocated memory on the host
  float* DevcData;  // Pointer to allocated memory on the GPU.  Note that the
                    //   host can know what the address of memory on the GPU
                    //   is, but it cannot simply de-reference that pointer
                    //   in host code.
};
typedef struct GpuMirroredFloat gpuFloat;

//-----------------------------------------------------------------------------
// ParticleSimulator: run a rudimentary simulation of particles
//-----------------------------------------------------------------------------
__global__ void ParticleSimulator()
{
  // Loop over all particles and compute the electrostatic potential.
  // Each thread will accumulate its own portion of the potential,
  // then pool the results at the end.
  int tidx = threadIdx.x;
  float qq = 0.0;
  while (tidx < cSh.nparticle) {

    // Still the naive way, to show how slow it is
    int i;
    for (i = 0; i < tidx; i++) {
      float dx = cSh.partX[tidx] - cSh.partX[i];
      float dy = cSh.partY[tidx] - cSh.partY[i];
      float dz = cSh.partZ[tidx] - cSh.partZ[i];
      float r = sqrt(dx*dx + dy*dy + dz*dz);
      qq += cSh.partQ[tidx] * cSh.partQ[i] / r;
    }

    // Increment counter
    tidx += blockDim.x;
  }

  // Accumulate energy
  atomicAdd(&cSh.Etot[0], qq);
}

//-----------------------------------------------------------------------------
// CreateGpuInt: constructor function for allocating memory in a gpuInt
//               instance.
//
// Arguments:
//   len:      the length of array to allocate
//   pin:      flag to have the memory pinned (non-pageable on the host side
//             for optimal transfer speed to the device)
//-----------------------------------------------------------------------------
gpuInt CreateGpuInt(int len, int pin)
{
  gpuInt G;

  G.len = len;
  G.IsPinned = pin;
  
  // Now that the official length is recorded, upgrade the real length
  // to the next convenient multiple of 128, so as to always allocate
  // GPU memory in 512-byte blocks.  This is for alignment purposes,
  // and keeping host to device transfers in line.
  len = ((len + 127) / 128) * 128;
  if (pin == 1) {
    hipHostAlloc((void **)&G.HostData, len * sizeof(int),
		  hipHostMallocMapped);
  }
  else {
    G.HostData = (int*)malloc(len * sizeof(int));
  }
  hipMalloc((void **)&G.DevcData, len * sizeof(int));
  memset(G.HostData, 0, len * sizeof(int));
  hipMemset((void *)G.DevcData, 0, len * sizeof(int));

  return G;
}

//-----------------------------------------------------------------------------
// DestroyGpuInt: destructor function for freeing memory in a gpuInt
//                instance.
//-----------------------------------------------------------------------------
void DestroyGpuInt(gpuInt *G)
{
  if (G->IsPinned == 1) {
    hipHostFree(G->HostData);
  }
  else {
    free(G->HostData);
  }
  hipFree(G->DevcData);
}

//-----------------------------------------------------------------------------
// UploadGpuInt: upload an integer array from the host to the device.
//-----------------------------------------------------------------------------
void UploadGpuInt(gpuInt *G)
{
  hipMemcpy(G->DevcData, G->HostData, G->len * sizeof(int),
             hipMemcpyHostToDevice);
}

//-----------------------------------------------------------------------------
// DownloadGpuInt: download an integer array from the host to the device.
//-----------------------------------------------------------------------------
void DownloadGpuInt(gpuInt *G)
{
  hipMemcpy(G->HostData, G->DevcData, G->len * sizeof(int),
	     hipMemcpyHostToDevice);
}

//-----------------------------------------------------------------------------
// CreateGpuFloat: constructor function for allocating memory in a gpuFloat
//                 instance.
//
// Arguments:
//   len:      the length of array to allocate
//   pin:      flag to have the memory pinned (non-pageable on the host side
//             for optimal transfer speed ot the device)
//-----------------------------------------------------------------------------
gpuFloat CreateGpuFloat(int len, int pin)
{
  gpuFloat G;

  G.len = len;
  G.IsPinned = pin;
  
  // Now that the official length is recorded, upgrade the real length
  // to the next convenient multiple of 128, so as to always allocate
  // GPU memory in 512-byte blocks.  This is for alignment purposes,
  // and keeping host to device transfers in line.
  len = ((len + 127) / 128) * 128;
  if (pin == 1) {
    hipHostAlloc((void **)&G.HostData, len * sizeof(float),
		  hipHostMallocMapped);
  }
  else {
    G.HostData = (float*)malloc(len * sizeof(float));
  }
  hipMalloc((void **)&G.DevcData, len * sizeof(float));
  memset(G.HostData, 0, len * sizeof(float));
  hipMemset((void *)G.DevcData, 0, len * sizeof(float));

  return G;
}

//-----------------------------------------------------------------------------
// DestroyGpuFloat: destructor function for freeing memory in a gpuFloat
//                  instance.
//-----------------------------------------------------------------------------
void DestroyGpuFloat(gpuFloat *G)
{
  if (G->IsPinned == 1) {
    hipHostFree(G->HostData);
  }
  else {
    free(G->HostData);
  }
  hipFree(G->DevcData);
}

//-----------------------------------------------------------------------------
// UploadGpuFloat: upload an float array from the host to the device.
//-----------------------------------------------------------------------------
void UploadGpuFloat(gpuFloat *G)
{
  hipMemcpy(G->DevcData, G->HostData, G->len * sizeof(float),
             hipMemcpyHostToDevice);
}

//-----------------------------------------------------------------------------
// DownloadGpuFloat: download an float array from the host to the device.
//-----------------------------------------------------------------------------
void DownloadGpuFloat(gpuFloat *G)
{
  hipMemcpy(G->HostData, G->DevcData, G->len * sizeof(float),
	     hipMemcpyHostToDevice);
}

//-----------------------------------------------------------------------------
// main
//-----------------------------------------------------------------------------
int main()
{
  int i, np;
  gpuInt particleTypes;
  gpuFloat particleXcoord, particleYcoord, particleZcoord, particleCharge;
  gpuFloat etot;
  
  // Create a small array of particles and populate it
  particleTypes  = CreateGpuInt(100000, 1);
  particleXcoord = CreateGpuFloat(100000, 1);
  particleYcoord = CreateGpuFloat(100000, 1);
  particleZcoord = CreateGpuFloat(100000, 1);
  particleCharge = CreateGpuFloat(100000, 1);

  // Allocate and initialize the total energy
  // accumulator on the host and on the device.
  etot = CreateGpuFloat(1, 1);
  
  // Initialize random number generator.  srand() SEEDS the generator,
  // thereafter each call to rand() will return a different number.
  // This is a reeally bad generator (much better methods with longer
  // periods before they start looping back over the same sequence are
  // available).
  srand(62052);
  
  // Place many, many particles
  np = 97913;
  for (i = 0; i < np; i++) {

    // Integer truncation would happen anyway, I'm just making it explicit
    particleTypes.HostData[i] = (int)(8 * rand());

    // Create some random coordinates (double-to-float conversion
    // is happening here.  On the GPU this can have performance
    // impact, so keep an eye on the data types at all times!
    particleXcoord.HostData[i] = 200.0 * (double)rand() / (double)RAND_MAX;
    particleYcoord.HostData[i] = 200.0 * (double)rand() / (double)RAND_MAX;
    particleZcoord.HostData[i] = 200.0 * (double)rand() / (double)RAND_MAX;
    particleCharge.HostData[i] =   0.5 - (double)rand() / (double)RAND_MAX;
  }

  // Show the CPU result
#if 0
  int j;
  double qq = 0.0;
  for (i = 0; i < np; i++) {
    for (j = 0; j < i; j++) {
      double dx = particleXcoord.HostData[i] - particleXcoord.HostData[j];
      double dy = particleYcoord.HostData[i] - particleYcoord.HostData[j];
      double dz = particleZcoord.HostData[i] - particleZcoord.HostData[j];
      double qfac = particleCharge.HostData[i] * particleCharge.HostData[j];
      qq += qfac / sqrt(dx*dx + dy*dy + dz*dz);
    }
  }
  printf("CPU result = %9.4lf\n", qq);
#endif
  
  // Stage critical constants--see cribSheet struct instance cSh above.
  cribSheet cnstage;
  cnstage.nparticle = np;
  cnstage.partX = particleXcoord.DevcData;
  cnstage.partY = particleYcoord.DevcData;
  cnstage.partZ = particleZcoord.DevcData;
  cnstage.partQ = particleCharge.DevcData;
  cnstage.Etot  = etot.DevcData; 
  
  // Upload all data to the device
  UploadGpuInt(&particleTypes);
  UploadGpuFloat(&particleXcoord);
  UploadGpuFloat(&particleYcoord);
  UploadGpuFloat(&particleZcoord);
  UploadGpuFloat(&particleCharge);

  // Upload the constants to the constants cache
  hipMemcpyToSymbol(HIP_SYMBOL(cSh), &cnstage, sizeof(cribSheet));  
  
  // Launch the kernel with different numbers of threads
  for (i = 1024; i >= 128; i /= 2) {

    // Zero the total energy and upload (this could be done by the GPU in
    // a separate kernel, but it's convenient enough to do it this way)
    etot.HostData[0] = 0.0;
    UploadGpuFloat(&etot);
    ParticleSimulator<<<1, i>>>();
  
    // Download the total energy
    DownloadGpuFloat(&etot);
    printf("Total energy (%4d threads) = %10.4f\n", i, etot.HostData[0]);
  }
  
  // Device synchronization
  hipDeviceSynchronize();
  
  return 0;
}
